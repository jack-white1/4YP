#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <float.h>


__global__ void axpy(int n, double a, double* x, double* y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n){y[i] = a*x[i] + y[i];}
}

__global__ void axpy(int n, float a, float* x, float* y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n){y[i] = a*x[i] + y[i];}
}

__global__ void axpy(int n, half a, half* x, half* y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	half b = __hmul(a,x[i]);
	if (i < n){y[i] = __hadd(b,y[i]);}
}

__global__ void axpy(int n, hip_bfloat16 a, hip_bfloat16* x, hip_bfloat16* y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	half b = __hmul(a,x[i]);
	if (i < n){y[i] = __hadd(b,y[i]);}
}


int main() {
	int N = 1<<25;
	int threadsPerBlock = 1024;
	double *dx, *dy, *d_dx, *d_dy;
	float *sx, *sy, *d_sx, *d_sy;
	half *hx, *hy, *d_hx, *d_hy;
	hip_bfloat16 *bx, *by, *d_bx, *d_by;

	hipEvent_t dstart, dstop, sstart, sstop, hstart, hstop, bstart, bstop;
	hipEvent_t dMemCpyH2DStart, dMemCpyH2DStop, sMemCpyH2DStart, sMemCpyH2DStop, hMemCpyH2DStart, hMemCpyH2DStop, bMemCpyH2DStart, bMemCpyH2DStop;
	hipEvent_t dMemCpyD2HStart, dMemCpyD2HStop, sMemCpyD2HStart, sMemCpyD2HStop, hMemCpyD2HStart, hMemCpyD2HStop, bMemCpyD2HStart, bMemCpyD2HStop;;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);

	hipEventCreate(&dstart);
	hipEventCreate(&dstop);
	hipEventCreate(&sstart);
	hipEventCreate(&sstop);
	hipEventCreate(&hstart);
	hipEventCreate(&hstop);
	hipEventCreate(&bstart);
	hipEventCreate(&bstop);

	hipEventCreate(&dMemCpyH2DStart);
	hipEventCreate(&dMemCpyH2DStop);
	hipEventCreate(&sMemCpyH2DStart);
	hipEventCreate(&sMemCpyH2DStop);
	hipEventCreate(&hMemCpyH2DStart);
	hipEventCreate(&hMemCpyH2DStop);
	hipEventCreate(&bMemCpyH2DStart);
	hipEventCreate(&bMemCpyH2DStop);

	hipEventCreate(&dMemCpyD2HStart);
	hipEventCreate(&dMemCpyD2HStop);
	hipEventCreate(&sMemCpyD2HStart);
	hipEventCreate(&sMemCpyD2HStop);
	hipEventCreate(&hMemCpyD2HStart);
	hipEventCreate(&hMemCpyD2HStop);
	hipEventCreate(&bMemCpyD2HStart);
	hipEventCreate(&bMemCpyD2HStop);


	dx = (double*)malloc(N*sizeof(double));
	dy = (double*)malloc(N*sizeof(double));
	sx = (float*)malloc(N*sizeof(float));
	sy = (float*)malloc(N*sizeof(float));
	hx = (half*)malloc(N*sizeof(half));
	hy = (half*)malloc(N*sizeof(half));
	bx = (hip_bfloat16*)malloc(N*sizeof(hip_bfloat16));
	by = (hip_bfloat16*)malloc(N*sizeof(hip_bfloat16));
	
	hipMalloc(&d_dx, N*sizeof(double));
	hipMalloc(&d_dy, N*sizeof(double));
	hipMalloc(&d_sx, N*sizeof(float));
	hipMalloc(&d_sy, N*sizeof(float));
	hipMalloc(&d_hx, N*sizeof(half));
	hipMalloc(&d_hy, N*sizeof(half));
	hipMalloc(&d_bx, N*sizeof(hip_bfloat16));
	hipMalloc(&d_by, N*sizeof(hip_bfloat16));

	
	srand (static_cast <unsigned> (time(0)));
	for (int i = 0; i< N; i++){
		dx[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
		dy[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
		sx[i] = static_cast <float> (dx[i]);
		sy[i] = static_cast <float> (dy[i]);
		hx[i] = __double2half(dx[i]);
		hy[i] = __double2half(dy[i]);
		bx[i] = __double2bfloat16(dx[i]);
		by[i] = __double2bfloat16(dy[i]);
		//printf ("%f, %f, %f, %f, %f, %f\n", dx[i], dy[i], sx[i], sy[i], __half2float(hx[i]), __half2float(hy[i]));
	}


	hipEventRecord(dMemCpyH2DStart);
	hipMemcpy(d_dx, dx, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_dy, dy, N*sizeof(double), hipMemcpyHostToDevice);
	hipEventRecord(dMemCpyH2DStop);

	hipEventRecord(sMemCpyH2DStart);
	hipMemcpy(d_sx, sx, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_sy, sy, N*sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(sMemCpyH2DStop);

	hipEventRecord(hMemCpyH2DStart);
	hipMemcpy(d_hx, hx, N*sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(d_hy, hy, N*sizeof(half), hipMemcpyHostToDevice);
	hipEventRecord(hMemCpyH2DStop);

	hipEventRecord(bMemCpyH2DStart);
	hipMemcpy(d_bx, bx, N*sizeof(hip_bfloat16), hipMemcpyHostToDevice);
	hipMemcpy(d_by, by, N*sizeof(hip_bfloat16), hipMemcpyHostToDevice);
	hipEventRecord(hMemCpyH2DStop);

	int numThreadBlocks = (N+threadsPerBlock-1)/threadsPerBlock;
	hipEventRecord(dstart);
	axpy<<<numThreadBlocks, threadsPerBlock>>>(N, static_cast <double> (2.0f), d_dx, d_dy);
	hipEventRecord(dstop);

	hipEventRecord(sstart);
	axpy<<<numThreadBlocks, threadsPerBlock>>>(N, 2.0f, d_sx, d_sy);
	hipEventRecord(sstop);

	hipEventRecord(hstart);
	axpy<<<numThreadBlocks, threadsPerBlock>>>(N, __float2half(2.0f), d_hx, d_hy);
	hipEventRecord(hstop);

	hipEventRecord(bstart);
	axpy<<<numThreadBlocks, threadsPerBlock>>>(N, __float2bfloat16(2.0f), d_bx, d_by);
	hipEventRecord(bstop);


	hipEventRecord(dMemCpyD2HStart);
	hipMemcpy(dy, d_dy, N*sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(dMemCpyD2HStop);

	hipEventRecord(sMemCpyD2HStart);
	hipMemcpy(sy, d_sy, N*sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(sMemCpyD2HStop);

	hipEventRecord(hMemCpyD2HStart);
	hipMemcpy(hy, d_hy, N*sizeof(half), hipMemcpyDeviceToHost);
	hipEventRecord(hMemCpyD2HStop);

	hipEventRecord(bMemCpyD2HStart);
	hipMemcpy(by, d_by, N*sizeof(hip_bfloat16), hipMemcpyDeviceToHost);
	hipEventRecord(bMemCpyD2HStop);

	double dmaxError = 0;
	double smaxError = 0;
	double hmaxError = 0;
	double bmaxError = 0;

	double dTotalError = 0;
	double sTotalError = 0;
	double hTotalError = 0;
	double bTotalError = 0;

	for (int i = 0; i < N; i++){
		double dError = abs(dy[i]-dy[i]);
		double sError = abs((static_cast <double> (sy[i]))-dy[i]);
		double hError = abs(static_cast <double> (__half2float(hy[i])) - dy[i]);
		double bError = abs(static_cast <double> (__bfloat162float(by[i])) - dy[i]);
		dmaxError = max(dmaxError, dError);
		smaxError = max(smaxError, sError);
		hmaxError = max(hmaxError, hError);
		bmaxError = max(bmaxError, bError);
		dTotalError += dError;
		sTotalError += sError;
		hTotalError += hError;
		bTotalError += bError;
		//printf("%lg, %f, %f\n", dy[i], sy[i], __half2float(hy[i]));
	}

	hipEventSynchronize(dstop);

	double dAvgError = dTotalError/N;
	double sAvgError = sTotalError/N;
	double hAvgError = hTotalError/N;
	double bAvgError = bTotalError/N;

	float d_H2D_milliseconds = 0;
	float s_H2D_milliseconds = 0;
	float h_H2D_milliseconds = 0;
	float b_H2D_milliseconds = 0;

	float d_compute_milliseconds = 0;
	float s_compute_milliseconds = 0;
	float h_compute_milliseconds = 0;
	float b_compute_milliseconds = 0;

	float d_D2H_milliseconds = 0;
	float s_D2H_milliseconds = 0;
	float h_D2H_milliseconds = 0;
	float b_D2H_milliseconds = 0;

	hipEventElapsedTime(&d_H2D_milliseconds, dMemCpyH2DStart, dMemCpyH2DStop);
	hipEventElapsedTime(&s_H2D_milliseconds, sMemCpyH2DStart, sMemCpyH2DStop);
	hipEventElapsedTime(&h_H2D_milliseconds, hMemCpyH2DStart, hMemCpyH2DStop);
	hipEventElapsedTime(&b_H2D_milliseconds, bMemCpyH2DStart, bMemCpyH2DStop);

	hipEventElapsedTime(&d_compute_milliseconds, dstart, dstop);
	hipEventElapsedTime(&s_compute_milliseconds, sstart, sstop);
	hipEventElapsedTime(&h_compute_milliseconds, hstart, hstop);
	hipEventElapsedTime(&b_compute_milliseconds, bstart, bstop);

	hipEventElapsedTime(&d_D2H_milliseconds, dMemCpyD2HStart, dMemCpyD2HStop);
	hipEventElapsedTime(&s_D2H_milliseconds, sMemCpyD2HStart, sMemCpyD2HStop);
	hipEventElapsedTime(&h_D2H_milliseconds, hMemCpyD2HStart, hMemCpyD2HStop);
	hipEventElapsedTime(&b_D2H_milliseconds, bMemCpyD2HStart, bMemCpyD2HStop);

	printf("-------------------------------------------------------------------------\n");
	printf("  Random Number A*X+Y on %d elements\n\n", N);
	printf("  %s\n",prop.name);
	printf("-------------------------------------------------------------------------\n");
	printf("|        | max error | avg error |  compute  | memcpy time | memcpy time |\n");
	printf("|  type  | to double | to double |    /ms    |  H2D /ms    |   D2H /ms   |\n");
	printf("-------------------------------------------------------------------------\n");
	printf("|        |           |           |           |             |             |\n");
	printf("|  fp64  |    N/A    |    N/A    | %6.2f    |   %6.2f    |   %6.2f    |\n", d_compute_milliseconds, d_H2D_milliseconds, d_D2H_milliseconds);
	printf("|        |           |           |           |             |             |\n");
	printf("-------------------------------------------------------------------------\n");
	printf("|        |           |           |           |             |             |\n");
	printf("|  fp32  | %1.3e | %1.3e | %6.2f    |   %6.2f    |   %6.2f    |\n", smaxError, sAvgError, s_compute_milliseconds, s_H2D_milliseconds, s_D2H_milliseconds);
	printf("|        |           |           |           |             |             |\n");
	printf("-------------------------------------------------------------------------\n");
	printf("|        |           |           |           |             |             |\n");
	printf("|  fp16  | %1.3e | %1.3e | %6.2f    |   %6.2f    |   %6.2f    |\n", hmaxError, hAvgError, h_compute_milliseconds, h_H2D_milliseconds, h_D2H_milliseconds);
	printf("|        |           |           |           |             |             |\n");
	printf("-------------------------------------------------------------------------\n");
	printf("|        |           |           |           |             |             |\n");
	printf("|  bf16  | %1.3e | %1.3e | %6.2f    |   %6.2f    |   %6.2f    |\n", bmaxError, bAvgError, b_compute_milliseconds, b_H2D_milliseconds, b_D2H_milliseconds);
	printf("|        |           |           |           |             |             |\n");
	printf("-------------------------------------------------------------------------\n");
	//printf("Max error between double, double: %1.5e, avg:%1.5e, took: %fms\n", dmaxError, dAvgError, d_compute_milliseconds);
	//printf("Max error between single, double: %1.5e, avg:%1.5e, took: %fms\n", smaxError, sAvgError, s_compute_milliseconds);
	//printf("Max error between half,   double: %1.5e, avg:%1.5e, took: %fms\n", hmaxError, hAvgError, h_compute_milliseconds);

	hipFree(d_dx);
	hipFree(d_dy);
	hipFree(d_sx);
	hipFree(d_sy);
	hipFree(d_hx);
	hipFree(d_hy);
	hipFree(d_bx);
	hipFree(d_by);
	free(dx);
	free(dy);
	free(sx);
	free(sy);
	free(hx);
	free(hy);
	free(bx);
	free(by);

}